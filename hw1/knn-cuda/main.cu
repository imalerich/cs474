#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>

// Provided data set.

#define RED		0
#define GREEN	1

const char * LABELS[] = {
	"Red",
	"Green"
};

const int d = 3; // 3-dimmensional data set
const int n = 6; // with 6 elements.

const float Y[] = {0, 0, 0};
const float X[] = {
	2, 3, 0,	// 1
	2, 0, 1,	// 2
	0, 1, 3,	// 3
	0, 1, 2,	// 4
	-1, 0, 1,	// 5
	1, -1, 1	// 6
};
const int L[] = {
	RED,		// 1
	RED,		// 2
	RED,		// 3
	GREEN,		// 4
	GREEN,		// 5
	RED			// 6
};

/**
 * Computes the Euclidean distance between the input vector 'Y' and all
 * vectors in the array 'X'.
 * An array of size 'n' containing each distance will be returned on completion.
 * \param n		Number of entries in the data set 'X'.
 * \param d		Dimension of each vector.
 * \param Y		Input vector to compare against 'X' (array of size d).
 * \param X		Input list of vectors (array of size n * d).
 * \param DIST	Output array of distances (array of size n).
 */
__global__ void calcDistE(int n, int d, float * Y, float * X, float * DIST) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n) { return; } // Index is not in bounds of the data set.

	float * x = &X[idx * d];
	float sum = 0.0f;

	// Compute the sum of squares over each dimmension...
	for (int i=0; i<d; ++i) {
		sum += pow(x[i] - Y[i], 2.0f);
	}

	// The distance will be the square root of that value.
	DIST[idx] = sqrt(sum);
}

/**
 * Sorts the input label array based on the input distances.
 * This is done with a parallel implementation of a selection sort,
 * which is fairly slow for a parallel sort, but should be faster 
 * than most serial sorts on large data sets.
 * \param n		The number of points in our data set.
 * \param DIST	Array if distances associated with our labels (array of size n).
 * \param L		List of class labels associated with distances (arry of size n).
 * \param OUT	Output copy of list L, sorted by correpsonding distances (array of size n).
 */
__global__ void findNearest(int n, float * DIST, int * L, int * OUT) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n) { return; } // Index is not in bounds of the data set.

	// This is the value we are sorting.
	float d = DIST[idx];
	int k = L[idx];

	// How many distances are smaller than the current one?
	int smaller = 0;
	for (int i=0; i<n; ++i) {
		if ((DIST[i] < d) ||
				// Break ties by index.
				(DIST[i] == d && i < idx)) {
			++smaller;
		}
	}

	// Set the output label.
	OUT[smaller] = k;
}

/**
 * Performs KNN on the given data.
 * \param n		Number of entries in the data set 'X'.
 * \param d		Dimension of each vector.
 * \param Y		Input vector to compare against 'X' (array of size d).
 * \param X		Input list of vectors (array of size n * d).
 * \param L		Labels coresponding to each of item of X (array of size n).
 * \param C		The number of classes that are valid in L.
 * \param Cstr	String representations for each class label (array of C strings).
 * \param k		How many neighbors to consider when making our assignment?
 */
void knn(const int n, const int d, const float * Y, 
		const float * X, const int * L, 
		const int C, const char ** Cstr, 
		const int k) {

	// Allocate the GPU arrays.
	float * cu_X, * cu_Y, * cu_DIST;
	int * cu_L, * cu_OUT;
	hipMalloc(&cu_X, n * d * sizeof(float));
	hipMalloc(&cu_Y, d * sizeof(float));
	hipMalloc(&cu_DIST, n * sizeof(float));
	hipMalloc(&cu_L, n * sizeof(int));
	hipMalloc(&cu_OUT, n * sizeof(int));

	// Copy the provided data to the GPU.
	hipMemcpy(cu_X, X, n * d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cu_Y, Y, d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cu_L, L, n * sizeof(int), hipMemcpyHostToDevice);

	// Compute distances in parallel in blocks of 256 threads.
	calcDistE<<<(n + 255)/256, 256>>>(n, d, cu_Y, cu_X, cu_DIST);
	findNearest<<<(n + 255)/256, 256>>>(n, cu_DIST, cu_L, cu_OUT);

	// Copy the sorted labels from our output
	int * OUT = (int *)malloc(n * sizeof(int));
	hipMemcpy(OUT, cu_OUT, n * sizeof(int), hipMemcpyDeviceToHost);

	// Count each class.
	int * count = (int *)malloc(C * sizeof(int));
	memset(count, 0, C * sizeof(int));
	for (int i=0; i<k; ++i) { ++count[OUT[i]]; }

	// Print the results.
	printf("knn: k=%d\n", k);
	for (int i=0; i<C; ++i) {
		printf("class %s:\t%d/%d\n", Cstr[i], count[i], k);
	}
	printf("\n");

	// Cleanup GPU...
	hipFree(cu_X);
	hipFree(cu_Y);
	hipFree(cu_DIST);
	hipFree(cu_L);
	hipFree(cu_OUT);

	// Cleanup CPU...
	free(OUT);
	free(count);
}

int main(int argc, const char ** argv) {
	knn(n, d, Y, X, L, 2, LABELS, 1);
	knn(n, d, Y, X, L, 2, LABELS, 3);
}
