#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>

#define RED		0
#define GREEN	1

/**
 * Computes the Euclidean distance between the input vector 'Y' and all
 * vectors in the array 'X'.
 * An array of size 'n' containing each distance will be returned on completion.
 * \param n Number of entries in the data set 'X'.
 * \param d Dimension of each vector.
 * \param Y Input vector to compare against 'X' (array of size d).
 * \param X Input list of vectors (array of size n * d).
 * \param D Output array of distances (array of size n).
 */
__global__ void calcDistE(int n, int d, float * Y, float * X, float * DIST) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= n) { return; } // Index is not in bounds of the data set.

	float * x = &X[idx * d];
	float sum = 0.0f;

	// Compute the sum of squares over each dimmension...
	for (int i=0; i<d; ++i) {
		sum += pow(x[i] - Y[i], 2.0f);
	}

	// The distance will be the square root of that value.
	DIST[idx] = sqrt(sum);
}

/**
 * Performs KNN on the given data.
 * \param n Number of entries in the data set 'X'.
 * \param d Dimension of each vector.
 * \param Y Input vector to compare against 'X' (array of size d).
 * \param X Input list of vectors (array of size n * d).
 * \param L Labels coresponding to each of item of X (array of size n).
 * \param C The number of classes that are valid in L.
 * \param k How many neighbors to consider when making our assignment?
 */
void knn(int n, int d, float * Y, float * X, int * L, int C, int k) {
	// Allocate the GPU arrays.
	float * cu_X, * cu_Y, * cu_DIST;
	hipMalloc(&cu_X, n * d * sizeof(float));
	hipMalloc(&cu_Y, d * sizeof(float));
	hipMalloc(&cu_DIST, n * sizeof(float));

	// Copy the provided data to the GPU.
	hipMemcpy(cu_X, X, n * d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cu_Y, Y, d * sizeof(float), hipMemcpyHostToDevice);

	// Compute distances in parallel in 1 block of 256 threads.
	calcDistE<<<(n + 255)/256, 256>>>(n, d, cu_Y, cu_X, cu_DIST);

	// Copy the distances back from the GPU to the CPU.
	float * dist = (float *)malloc(n * sizeof(float));
	hipMemcpy(dist, cu_DIST, n * sizeof(float), hipMemcpyDeviceToHost);

	// Count the number of occurences of each label.
	int * labels = (int *)malloc(C * sizeof(int));
	for (int i=0; i<C; ++i) { labels[i] = 0; }

	// This is horribly ineficient [O(n^2) when k=n].
	// would be much better to keep cu_DIST on the GPU
	// (and associate labels with it), sort it there
	// then just read data from that array.
	for (int i=0; i<k; i++) {
		int least = -1;
		for (int j=0; j<n; j++) {
			if (least < 0 || dist[j] < dist[least]) {
				least = j;
			}
		}

		dist[least] = FLT_MAX; // Ignore this value on the next pass.
		++labels[ L[least] ]; // Increment the label given for this element.
	}

	// Print the results.
	printf("knn: k=%d\n", k);
	for (int i=0; i<C; i++) {
		printf("class %d:\t%d/%d\n", i, labels[i], k);
	}
	printf("\n");

	// Cleanup...
	hipFree(cu_X);
	hipFree(cu_Y);
	hipFree(cu_DIST);
	free(dist);
	free(labels);
}

int main(int argc, const char ** argv) {
	const int d = 3; // 3-dimmensional data set
	const int n = 6; // with 6 elements.

	float Y[] = {0, 0, 0};
	float X[] = {
		2, 3, 0,	// 1
		2, 0, 1,	// 2
		0, 1, 3,	// 3
		0, 1, 2,	// 4
		-1, 0, 1,	// 5
		1, -1, 1	// 6
	};
	int L[] = {
		RED,		// 1
		RED,		// 2
		RED,		// 3
		GREEN,		// 4
		GREEN,		// 5
		RED			// 6
	};

	// Run k-nearest neighbors for k=1 and k=3.
	knn(n, d, Y, X, L, 2, 1);
	knn(n, d, Y, X, L, 2, 3);
}
